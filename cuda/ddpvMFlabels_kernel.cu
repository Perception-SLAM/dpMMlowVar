#include "hip/hip_runtime.h"
/* Copyright (c) 2015, Julian Straub <jstraub@csail.mit.edu> Licensed
 * under the MIT license. See the license file LICENSE.
 */

#include <stdio.h>
#include <jsCore/cuda_global.h>

#define DIM 3
#include <dpMMlowVar/ddpvMF_cuda_helper.h>
// executions per thread
#define N_PER_T 16
#define BLOCK_SIZE 256


template<typename T, uint32_t K, uint32_t BLK_SIZE>
__global__ void ddpvMFlabelAssign_kernel(T *d_q, T *d_p, uint32_t *z,
    uint32_t *d_Ns, T *d_ages, T *d_ws, T lambda, T beta, T Q, uint32_t *d_iAction,
    uint32_t i0, uint32_t N)
{
  __shared__ T p[DIM*(K+1)]; // K+1 because K might be 0 and 0 size arrays are not appreciated
//  __shared__ T ages[K+1];
  __shared__ T Ns[K+1];
//  __shared__ T ws[K+1];
  __shared__ uint32_t iAction[BLK_SIZE]; // id of first action (revieval/new) for one core

  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // caching and init
  iAction[tid] = UNASSIGNED;
  if(tid < DIM*K) p[tid] = d_p[tid];
//  if(tid < K) ages[tid] = d_ages[tid];
//  if (K>=1) return;
  if(tid < K) Ns[tid] = d_Ns[tid];
//  if(tid < K) ws[tid] = d_ws[tid];
  __syncthreads(); // make sure that ys have been cached


  for(int id=idx*N_PER_T; id<min(N,(idx+1)*N_PER_T); ++id)
  {
    uint32_t z_i = K;
    T sim_closest = lambda + 1.;
    T sim_k = 0.;
    T* p_k = p;
    T q_i[DIM];
    q_i[0] = d_q[id*DIM];
    q_i[1] = d_q[id*DIM+1];
    q_i[2] = d_q[id*DIM+2];
    if (q_i[0]!=q_i[0] || q_i[1]!=q_i[1] || q_i[2]!=q_i[2])
    {
      // normal is nan -> break out here
      z[id] = UNASSIGNED;
    }else{
      for (uint32_t k=0; k<K; ++k)
      {
        T dot = min(1.0,max(-1.0,q_i[0]*p_k[0] + q_i[1]*p_k[1]
              + q_i[2]*p_k[2]));
        T zeta = acos(dot);

        if(Ns[k] == 0)
        {// cluster not instantiated yet in this timestep
          T age = d_ages[k]; // TODO d_ages size is not always = K
          //TODO: using small angle approximation here!
          sim_k = distToUninstantiatedSmallAngleApprox<T>(zeta,
              age,beta,d_ws[k],Q);
//          sim_k = distToUninstantiated<T,10>(zeta,age,beta,d_ws[k],Q,1e-6);
        }else{ // cluster instantiated
          sim_k = dot;
        }
        if(sim_k > sim_closest)
        {
          sim_closest = sim_k;
          z_i = k;
        }
        p_k += DIM;
      }
      if (z_i == K || Ns[z_i] == 0)
      {
        iAction[tid] = id;
        break; // save id at which an action occured and break out because after
        // that id anything more would be invalid.
      }
      z[id] = z_i;
    }
  }

  // min() reduction
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s)
    {
      iAction[tid] = min(iAction[tid], iAction[s+tid]);
    }
    __syncthreads();
  }
  if(tid == 0) {
    // reduce the last two remaining matrixes directly into global memory
    atomicMin(d_iAction, min(iAction[0],iAction[1]));
  }
};


void ddpvMFlabels_gpu( double *d_q,  double *d_p,  uint32_t *d_z,
    uint32_t *d_Ns, double *d_ages, double *d_ws, double lambda, double beta,
    double Q, uint32_t k0, uint32_t K, uint32_t i0, uint32_t N, uint32_t *d_iAction)
{
  const uint32_t BLK_SIZE = BLOCK_SIZE/2;
  assert(BLK_SIZE > DIM*K+DIM*(DIM-1)*K);

  dim3 threads(BLK_SIZE,1,1);
  dim3 blocks(N/(BLK_SIZE*N_PER_T)+(N%(BLK_SIZE*N_PER_T)>0?1:0),1,1);
  if(K == 0){
    *d_iAction =0;
  }else if(K == 1){
    ddpvMFlabelAssign_kernel<double,1,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==2){
    ddpvMFlabelAssign_kernel<double,2,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==3){
    ddpvMFlabelAssign_kernel<double,3,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==4){
    ddpvMFlabelAssign_kernel<double,4,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==5){
    ddpvMFlabelAssign_kernel<double,5,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==6){
    ddpvMFlabelAssign_kernel<double,6,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==7){
    ddpvMFlabelAssign_kernel<double,7,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==8){
    ddpvMFlabelAssign_kernel<double,8,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==9){
    ddpvMFlabelAssign_kernel<double,9,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==10){
    ddpvMFlabelAssign_kernel<double,10,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==11){
    ddpvMFlabelAssign_kernel<double,11,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==12){
    ddpvMFlabelAssign_kernel<double,12,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==13){
    ddpvMFlabelAssign_kernel<double,13,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==14){
    ddpvMFlabelAssign_kernel<double,14,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==15){
    ddpvMFlabelAssign_kernel<double,15,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==16){
    ddpvMFlabelAssign_kernel<double,16,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else{
    assert(false);
  }
  checkCudaErrors(hipDeviceSynchronize());
};


void ddpvMFlabels_gpu( float *d_q,  float *d_p,  uint32_t *d_z,
    uint32_t *d_Ns, float *d_ages, float *d_ws, float lambda, float beta,
    float Q, uint32_t k0, uint32_t K, uint32_t i0, uint32_t N, uint32_t *d_iAction)
{
  const uint32_t BLK_SIZE = BLOCK_SIZE;
  assert(BLK_SIZE > DIM*K+DIM*(DIM-1)*K);

  dim3 threads(BLK_SIZE,1,1);
  dim3 blocks(N/(BLK_SIZE*N_PER_T)+(N%(BLK_SIZE*N_PER_T)>0?1:0),1,1);
  if(K == 0){
    *d_iAction = 0;
  }else if(K == 1){
    ddpvMFlabelAssign_kernel<float,1,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==2){
    ddpvMFlabelAssign_kernel<float,2,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==3){
    ddpvMFlabelAssign_kernel<float,3,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==4){
    ddpvMFlabelAssign_kernel<float,4,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==5){
    ddpvMFlabelAssign_kernel<float,5,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==6){
    ddpvMFlabelAssign_kernel<float,6,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==7){
    ddpvMFlabelAssign_kernel<float,7,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==8){
    ddpvMFlabelAssign_kernel<float,8,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==9){
    ddpvMFlabelAssign_kernel<float,9,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==10){
    ddpvMFlabelAssign_kernel<float,10,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==11){
    ddpvMFlabelAssign_kernel<float,11,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==12){
    ddpvMFlabelAssign_kernel<float,12,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==13){
    ddpvMFlabelAssign_kernel<float,13,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==14){
    ddpvMFlabelAssign_kernel<float,14,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==15){
    ddpvMFlabelAssign_kernel<float,15,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else if(K==16){
    ddpvMFlabelAssign_kernel<float,16,BLK_SIZE><<<blocks,threads>>>(
        d_q, d_p, d_z, d_Ns, d_ages, d_ws, lambda, beta, Q, d_iAction, i0, N);
  }else{
    assert(false);
  }
  checkCudaErrors(hipDeviceSynchronize());
};

