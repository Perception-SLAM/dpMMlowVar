#include "hip/hip_runtime.h"

#include <stdio.h>
#include <float.h>
#include <dpMMlowVar/cuda_global.h>

// executions per thread
#define K_MAX 50
#define N_PER_T 16
#define BLOCK_SIZE 256

template<typename T>
__device__ inline T distToUninstantiated( T distsq, T age, T w, T Q, T tau, T thresh)
{
  return Q*age+1.0/(1.0+1.0/w+age*tau)*distsq;
}

template<typename T, uint32_t BLK_SIZE>
__global__ void ddpLabelAssignSpecial_kernel(T *d_q, T *d_oldp, T
    *d_ages, T *d_ws, T lambda, T Q, T tau, uint32_t *d_asgnIdces,
    uint32_t N, uint32_t K)
{
  //__shared__ T oldp[DIM*K];
  __shared__ uint32_t asgnIdces[K_MAX*BLK_SIZE]; //for each thread, index selected for each old K
  __shared__ T oldp[K_MAX*DIM];

  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // caching and init
  for(int k = 0; k < K; k++){
    asgnIdces[K*tid+k] = UNASSIGNED;
    if(tid < DIM) oldp[k*DIM+tid] = d_oldp[k*DIM+tid];
  }
  __syncthreads(); // make sure that ys have been cached

  for(int id=idx*N_PER_T; id<min(N,(idx+1)*N_PER_T); ++id)
  {
    T max_sim_k = FLT_MAX;
    uint32_t max_k = UNASSIGNED;
    T sim_k = 0.;
    T* p_k = oldp;
    T q_i[DIM];
    q_i[0] = d_q[id*DIM];
    q_i[1] = d_q[id*DIM+1];
    q_i[2] = d_q[id*DIM+2];
    if (q_i[0] ==q_i[0] && q_i[1] ==q_i[1] && q_i[2]==q_i[2])// only do this for q not nan
    {
      for (uint32_t k=0; k<K; ++k) {
        T distsq = (q_i[0] - p_k[0])*(q_i[0] - p_k[0])
        		 +(q_i[1] - p_k[1])*(q_i[1] - p_k[1])
        		 +(q_i[2] - p_k[2])*(q_i[2] - p_k[2]);
        sim_k = distToUninstantiated<T>(distsq,d_ages[k],d_ws[k],Q,tau,1e-6);
        if(sim_k < lambda && max_sim_k > sim_k)
        {
          max_sim_k = sim_k;
          max_k = k;
        }
        p_k += DIM;
      }
      if(max_k < K && id < asgnIdces[K*tid+max_k]){
        asgnIdces[K*tid+max_k] = id;
      }
    }
  }
  // min() reduction
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s)
    {
      for(uint32_t k = 0; k < K; ++k){
        if(asgnIdces[K*tid+k] > asgnIdces[K*(s+tid)+k]){
          asgnIdces[K*tid+k] = asgnIdces[K*(s+tid)+k];
        } 
      }
    }
    __syncthreads();
  }

  //reduce the 2 remaining into the output d_asgnCosts/d_asgnIndices
  if(tid < K) {
    if(asgnIdces[tid] < asgnIdces[K+tid]){
      // leads to the smallest index of minimal cost value (but only minimal wrt to its block)
      // this is not the argmin over all values - that is probably not possible atomically
      atomicMin(&d_asgnIdces[tid], asgnIdces[tid]);
    } else {
      atomicMin(&d_asgnIdces[tid], asgnIdces[K+tid]);
    }
  }

};

template<typename T, uint32_t K, uint32_t BLK_SIZE>
__global__ void ddpLabelAssignSpecial_kernel(T *d_q, T *d_oldp, T *d_ages, T *d_ws, T lambda, T Q, T tau, uint32_t *d_asgnIdces, uint32_t N)
{
  //__shared__ T oldp[DIM*K];
  __shared__ uint32_t asgnIdces[K*BLK_SIZE]; //for each thread, index selected for each old K
  __shared__ T oldp[K*DIM];

  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // caching and init
  for(int k = 0; k < K; k++){
    asgnIdces[K*tid+k] = UNASSIGNED;
    if(tid < DIM) oldp[k*DIM+tid] = d_oldp[k*DIM+tid];
  }
  __syncthreads(); // make sure that ys have been cached

  for(int id=idx*N_PER_T; id<min(N,(idx+1)*N_PER_T); ++id)
  {
    T max_sim_k = FLT_MAX;
    uint32_t max_k = UNASSIGNED;
    T sim_k = 0.;
    T* p_k = oldp;
    T q_i[DIM];
    q_i[0] = d_q[id*DIM];
    q_i[1] = d_q[id*DIM+1];
    q_i[2] = d_q[id*DIM+2];
    if (q_i[0] ==q_i[0] && q_i[1] ==q_i[1] && q_i[2]==q_i[2])// only do this for q not nan
    {
      for (uint32_t k=0; k<K; ++k) {
        T distsq = (q_i[0] - p_k[0])*(q_i[0] - p_k[0])
        		 +(q_i[1] - p_k[1])*(q_i[1] - p_k[1])
        		 +(q_i[2] - p_k[2])*(q_i[2] - p_k[2]);
        sim_k = distToUninstantiated<T>(distsq,d_ages[k],d_ws[k],Q,tau,1e-6);
        if(sim_k < lambda && max_sim_k > sim_k)
        {
          max_sim_k = sim_k;
          max_k = k;
        }
        p_k += DIM;
      }
      if(max_k < K && id < asgnIdces[K*tid+max_k]){
        asgnIdces[K*tid+max_k] = id;
      }
    }
  }
  // min() reduction
  __syncthreads(); //sync the threads
#pragma unroll
  for(int s=(BLK_SIZE)/2; s>1; s>>=1) {
    if(tid < s)
    {
      for(uint32_t k = 0; k < K; ++k){
        if(asgnIdces[K*tid+k] > asgnIdces[K*(s+tid)+k]){
          asgnIdces[K*tid+k] = asgnIdces[K*(s+tid)+k];
        } 
      }
    }
    __syncthreads();
  }

  //reduce the 2 remaining into the output d_asgnCosts/d_asgnIndices
  if(tid < K) {
    if(asgnIdces[tid] < asgnIdces[K+tid]){
      // leads to the smallest index of minimal cost value (but only minimal wrt to its block)
      // this is not the argmin over all values - that is probably not possible atomically
      atomicMin(&d_asgnIdces[tid], asgnIdces[tid]);
    } else {
      atomicMin(&d_asgnIdces[tid], asgnIdces[K+tid]);
    }
  }

};

extern void ddpLabelsSpecial_gpu( double *d_q,  double *d_oldp, double *d_ages, double *d_ws, double lambda, double Q, 
    double tau, uint32_t K, uint32_t N, uint32_t *d_asgnIdces)
{
  const uint32_t BLK_SIZE = BLOCK_SIZE/2;
  assert(K >= 1);//only run the special kernel if there is at least one old cluster
  assert(BLK_SIZE > DIM*K+DIM*(DIM-1)*K);

  dim3 threads(BLK_SIZE,1,1);
  dim3 blocks(N/(BLK_SIZE*N_PER_T)+(N%(BLK_SIZE*N_PER_T)>0?1:0),1,1);
  if(K == 1){
    ddpLabelAssignSpecial_kernel<double,1, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==2){
    ddpLabelAssignSpecial_kernel<double,2, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==3){
    ddpLabelAssignSpecial_kernel<double,3, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==4){
    ddpLabelAssignSpecial_kernel<double,4, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==5){
    ddpLabelAssignSpecial_kernel<double,5, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==6){
    ddpLabelAssignSpecial_kernel<double,6, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==7){
    ddpLabelAssignSpecial_kernel<double,7, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==8){
    ddpLabelAssignSpecial_kernel<double,8, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==9){
    ddpLabelAssignSpecial_kernel<double,9, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==10){
    ddpLabelAssignSpecial_kernel<double,10, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==11){
    ddpLabelAssignSpecial_kernel<double,11, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces,  N);
  }else if(K==12){
    ddpLabelAssignSpecial_kernel<double,12, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==13){
    ddpLabelAssignSpecial_kernel<double,13, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==14){
    ddpLabelAssignSpecial_kernel<double,14, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==15){
    ddpLabelAssignSpecial_kernel<double,15, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==16){
    ddpLabelAssignSpecial_kernel<double,16, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else{
    ddpLabelAssignSpecial_kernel<double,BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N, K);
  }
  checkCudaErrors(hipDeviceSynchronize());

};


extern void ddpLabelsSpecial_gpu( float *d_q,  float *d_oldp, float *d_ages,
    float *d_ws, float lambda, float Q, float tau, uint32_t K, uint32_t N,
    uint32_t *d_asgnIdces)
{
  const uint32_t BLK_SIZE = BLOCK_SIZE/2;
  assert(K >= 1);//only run the special kernel if there is at least one old cluster
  assert(BLK_SIZE > DIM*K+DIM*(DIM-1)*K);

  dim3 threads(BLK_SIZE,1,1);
  dim3 blocks(N/(BLK_SIZE*N_PER_T)+(N%(BLK_SIZE*N_PER_T)>0?1:0),1,1);
  if(K == 1){
    ddpLabelAssignSpecial_kernel<float,1, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==2){
    ddpLabelAssignSpecial_kernel<float,2, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==3){
    ddpLabelAssignSpecial_kernel<float,3, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==4){
    ddpLabelAssignSpecial_kernel<float,4, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==5){
    ddpLabelAssignSpecial_kernel<float,5, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==6){
    ddpLabelAssignSpecial_kernel<float,6, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==7){
    ddpLabelAssignSpecial_kernel<float,7, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==8){
    ddpLabelAssignSpecial_kernel<float,8, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==9){
    ddpLabelAssignSpecial_kernel<float,9, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==10){
    ddpLabelAssignSpecial_kernel<float,10, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==11){
    ddpLabelAssignSpecial_kernel<float,11, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces,  N);
  }else if(K==12){
    ddpLabelAssignSpecial_kernel<float,12, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==13){
    ddpLabelAssignSpecial_kernel<float,13, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==14){
    ddpLabelAssignSpecial_kernel<float,14, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==15){
    ddpLabelAssignSpecial_kernel<float,15, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else if(K==16){
    ddpLabelAssignSpecial_kernel<float,16, BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N);
  }else{
    ddpLabelAssignSpecial_kernel<float,BLK_SIZE><<<blocks, threads>>>(
        d_q, d_oldp, d_ages, d_ws, lambda, Q, tau, d_asgnIdces, N, K);
  }
  checkCudaErrors(hipDeviceSynchronize());

};

